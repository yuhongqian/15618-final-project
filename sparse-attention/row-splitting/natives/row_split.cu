#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <hipsparse.h>
#include "../../utils/cycleTimer.h"

#define DEVICE 0
#define MAX_SEQ_LEN 512

using namespace std;

static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

        case HIPSPARSE_STATUS_SUCCESS:
            return"HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return"HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return"HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return"HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return"HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return"HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return"HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return"HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return"HIPSPARSE_STATUS_ZERO_PIVOT";

        case HIPSPARSE_STATUS_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_NOT_SUPPORTED";
    }

    return"<unknown>";
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if (HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr,"CUSPARSE error in file '%s', line %d, error %s terminating!", __FILE__, __LINE__, _cusparseGetErrorEnum(err));
        assert(0);
    }
}
extern"C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }


__device__ __inline__ int device_get_idx_col(int row, int col, int height) {
    return col * height + row;
}

__device__ __inline__ int device_get_idx(int row, int col, int width) {
    return row * width + col;
}

__device__ __inline__ void print_arr(float* arr, int len) {
    for (int i = 0; i < len; i++) {
        printf("%f ", arr[i]);
    }
}

__device__ __inline__ void print_int_arr(int* arr, int len) {
    for (int i = 0; i < len; i++) {
        printf("%d ", arr[i]);
    }
}

// Currently, each thread is responsible for one output element
__global__ void device_spmm(int m, int k, int n, float *A_data, int *A_row_ptrs, int *A_col_indices,
                            float *B_dense, float *C_dense) {
    int m_idx = blockIdx.y;
    int n_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (m_idx >= m) return;

    // TODO: reduce shared memory size
    // Load the corresponding row from A into shared mem
    __shared__ float curr_row[MAX_SEQ_LEN];
    __shared__ int curr_B_row_idxs[MAX_SEQ_LEN];

    int row_start = A_row_ptrs[m_idx];
    int row_end = A_row_ptrs[m_idx + 1];
    int row_nnz = row_end - row_start;
    for (int i = threadIdx.x; i < row_nnz; i += blockDim.x) {
        curr_row[i] = A_data[row_start + i];
        curr_B_row_idxs[i] = A_col_indices[row_start + i];
    }
    __syncthreads();

    if (n_idx >= n) return;
    // Each thread loops through the corresponding col in B
    float res = 0;
    for (int i = 0; i < row_nnz; i++) {
        // Note A_dense and B_dense are col-major
        float elem = B_dense[device_get_idx_col(curr_B_row_idxs[i], n_idx, k)];
        res += curr_row[i] * elem;
    }
    C_dense[device_get_idx(m_idx, n_idx, n)] = res;
}

int get_grid_len(int number_elems, int block_dim) {
    return (number_elems + block_dim - 1) / block_dim;
}

void print_a_row(float *dense, int row, int width) {
    printf("row = %d, width = %d\n", row, width);
    for (int i = 0; i < width; i++) {
        printf("%f ", dense[row * width + i]);
    }
    printf("\n");
}

//void row_split_spmm(const float *h_A_dense, const float *h_B_dense, int m, int k, int n) {
void row_split_spmm(torch::Tensor *h_A_dense, torch::Tensor *h_B_dense, torch::Tensor *h_C_dense, int m, int k, int n) {

        int val;
        hipDeviceGetAttribute(&val, hipDeviceAttributePageableMemoryAccess, DEVICE);
        float *A_dense = h_A_dense.data_ptr();
        float *B_dense = h_B_dense.data_ptr();
        float *C_dense = h_C_dense.data_ptr();
//        gpuErrchk(hipMalloc(&A_dense, m * k * sizeof(float)));
//        gpuErrchk(hipMalloc(&B_dense, k * n * sizeof(float)));
//        gpuErrchk(hipMalloc(&C_dense, m * n * sizeof(float)));
//        gpuErrchk(hipMemcpy(A_dense, h_A_dense, m * k * sizeof(float), hipMemcpyHostToDevice));
//        gpuErrchk(hipMemcpy(B_dense, h_B_dense, k * n * sizeof(float), hipMemcpyHostToDevice));
        // --- Initialize cuSPARSE

        hipsparseHandle_t handle;
        cusparseSafeCall(hipsparseCreate(&handle));
        // Initialize matrix descriptors
        hipsparseMatDescr_t descrA;
        cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
        cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

        // Get nnz's
        int nnzA = 0;
        int *nnzPerVectorA;
        const int lda = m;
        gpuErrchk(hipMalloc(&nnzPerVectorA, m * sizeof(int)));
        cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, m, k, descrA, A_dense, lda, nnzPerVectorA, &nnzA));
        printf("nnzA = %d\n", nnzA);
        // declare CSR data
        float *A_data;
        gpuErrchk(hipMalloc(&A_data, nnzA * sizeof(float)));

        // declare CSR row-pointers & col indices
        int *A_row_ptrs, *A_col_indices;
        gpuErrchk(hipMalloc(&A_row_ptrs, (m + 1) * sizeof(int)));
        gpuErrchk(hipMalloc(&A_col_indices, nnzA * sizeof(int)));

        // fill CSR arrays
        cusparseSafeCall(hipsparseSdense2csr(handle, m, k, descrA, A_dense, lda, nnzPerVectorA, A_data, A_row_ptrs,
                                            A_col_indices));

        // invoke kernel
        dim3 blockDim(32);
        // TODO: reduce grid size using nnz

        dim3 gridDim(get_grid_len(n, blockDim.x), get_grid_len(m, blockDim.y));

        double start = CycleTimer::currentSeconds();
        device_spmm<<<gridDim, blockDim>>>(m, k, n, A_data, A_row_ptrs, A_col_indices, B_dense, C_dense);
        gpuErrchk(hipDeviceSynchronize());
        double end = CycleTimer::currentSeconds();
        printf("row-spliting matmul:    %.4f ms\n", 1000.f * (end - start));

        float *h_C_dense = (float*)malloc(m * n * sizeof(float));
        hipMemcpy(h_C_dense, C_dense, m * n * sizeof(float), hipMemcpyDeviceToHost);

        free(h_C_dense);
        hipFree(A_dense);
        hipFree(B_dense);
        hipFree(C_dense);
    }
}

