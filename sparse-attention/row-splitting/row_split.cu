#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "../utils/cycleTimer.h"

#define DEVICE 0
#define MAX_SEQ_LEN 512

static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

        case HIPSPARSE_STATUS_SUCCESS:
            return"HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return"HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return"HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return"HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return"HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return"HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return"HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return"HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return"HIPSPARSE_STATUS_ZERO_PIVOT";

        case HIPSPARSE_STATUS_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_NOT_SUPPORTED";
    }

    return"<unknown>";
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if (HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr,"CUSPARSE error in file '%s', line %d, error %s terminating!", __FILE__, __LINE__, _cusparseGetErrorEnum(err));
        assert(0);
    }
}
extern"C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

__device__ __inline__ int device_get_idx(int row, int col, int width) {
    return row * width + col;
}

// Currently, each thread is responsible for one output element
__global__ void device_spmm(int m, int k, int n, float *A_data, int *A_row_ptrs, int *A_col_indices,
                            const float *B_dense, float *C_dense) {
    int m_idx = blockIdx.y;
    int n_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (m_idx > m || n_idx > n) return;

    // TODO: reduce shared memory size
    // Load the corresponding row from A into shared mem
    __shared__ float curr_row[MAX_SEQ_LEN];
    __shared__ float curr_col_idxs[MAX_SEQ_LEN];

    // n < k, so each thread may be responsible for loading multiple row elements from A into shared mem
    int row_start = A_row_ptrs[m_idx];
    int row_end = A_row_ptrs[m_idx + 1];
    int row_nnz = row_end - row_start;
    for (int i = n_idx; i < row_nnz; i += blockDim.x) {
        curr_row[i] = A_data[row_start + i];
        curr_col_idxs[i] = A_col_indices[row_start + i];
    }
    __syncthreads();

    // Each thread loops through the corresponding col in B
    float res = 0;
    for (int i = 0; i < row_nnz; i++) {
        float elem = curr_row[i] * curr_col_idxs[i];
        res += curr_row[i] * elem;
    }
    C_dense[device_get_idx(m_idx, n_idx, n)];
}

int get_grid_len(int number_elems, int block_dim) {
    return (number_elems + block_dim - 1) / block_dim;
}

void row_split_spmm(const float *h_A_dense, const float *h_B_dense, int m, int k, int n) {
    int val;
    hipDeviceGetAttribute(&val, hipDeviceAttributePageableMemoryAccess, DEVICE);
    float *A_dense, *B_dense, *C_dense;
    gpuErrchk(hipMalloc(&A_dense, m * k * sizeof(float)));
    gpuErrchk(hipMalloc(&B_dense, k * n * sizeof(float)));
    gpuErrchk(hipMemcpy(A_dense, h_A_dense, m * k * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(B_dense, h_B_dense, k * n * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc(&C_dense, m * n * sizeof(float)));
    // --- Initialize cuSPARSE

    hipsparseHandle_t handle;
    cusparseSafeCall(hipsparseCreate(&handle));
    // Initialize matrix descriptors
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    // Get nnz's
    int nnzA = 0;
    int *nnzPerVectorA;
    const int lda = m;
    gpuErrchk(hipMalloc(&nnzPerVectorA, k * sizeof(*nnzPerVectorA)));
    cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, m, k, descrA, A_dense, lda, nnzPerVectorA, &nnzA));

    // declare CSR data
    float *A_data;
    gpuErrchk(hipMalloc(&A_data, nnzA * sizeof(*A_data)));

    // declare CSR row-pointers & col indices
    int *A_row_ptrs, *A_col_indices;
    gpuErrchk(hipMalloc(&A_row_ptrs, (m + 1) * sizeof(*A_row_ptrs)));
    gpuErrchk(hipMalloc(&A_col_indices, nnzA * sizeof(*A_col_indices)));

    // fill CSR arrays
    cusparseSafeCall(hipsparseSdense2csr(handle, m, k, descrA, A_dense, lda, nnzPerVectorA, A_data, A_row_ptrs,
                                        A_col_indices));

    // invoke kernel
    dim3 blockDim(32);   // each
    // TODO: reduce grid size using nnz
    dim3 gridDim(get_grid_len(m, blockDim.x), get_grid_len(n, blockDim.y));
    double start = CycleTimer::currentSeconds();
    device_spmm<<<gridDim, blockDim>>>(m, k, n, A_data, A_row_ptrs, A_col_indices, B_dense, C_dense);
    gpuErrchk(hipDeviceSynchronize());
    double end = CycleTimer::currentSeconds();
    printf("row-spliting matmul:    %.4f ms\n", 1000.f * (end - start));

}
