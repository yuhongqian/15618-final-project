#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

void row_split_spmm(const float *A_dense, const float *B_dense, int m, int k, int n) {
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Initialize matrix descriptors
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // Get nnz's
    int nnzA = 0;
    int *nnzPerVectorA;
    const int lda = m;
    hipMallocManaged(&nnzPerVectorA, k * sizeof(*nnzPerVectorA));
    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, m, k, descrA, A_dense, lda, nnzPerVectorA, &nnzA);

    // declare CSR data
    float *A_data;
    hipMallocManaged(&A_data, nnzA * sizeof(*A_data));

    // declare CSR row-pointers & col indices
    int *A_row_ptrs, *A_col_indices;
    hipMallocManaged(&A_row_ptrs, (m + 1) * sizeof(*A_row_ptrs));
    hipMallocManaged(&A_col_indices, nnzA * sizeof(*A_col_indices));

    // fill CSR arrays
    hipsparseSdense2csr(handle, m, k, descrA, A_dense, lda, nnzPerVectorA, A_data, A_row_ptrs, A_col_indices);

    // invoke kernel

}
