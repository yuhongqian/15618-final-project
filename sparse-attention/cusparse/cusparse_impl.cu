#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

        case HIPSPARSE_STATUS_SUCCESS:
            return"HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return"HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return"HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return"HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return"HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return"HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return"HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return"HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return"HIPSPARSE_STATUS_ZERO_PIVOT";

        case HIPSPARSE_STATUS_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_NOT_SUPPORTED";
    }

    return"<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if (HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr,"CUSPARSE error in file '%s', line %d, error %s terminating!", __FILE__, __LINE__, _cusparseGetErrorEnum(err));
        assert(0);
    }
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
extern"C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }


/********/
/* MAIN */
/********/


void cusparse_mmul(const float *h_A_dense, const float *h_B_dense, int m, int k, int n)
{
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;
    cusparseSafeCall(hipsparseCreate(&handle));

    /**************************/
    /* SETTING UP THE PROBLEM */
    /**************************/

    float *h_C_dense = (float*)malloc(m * n * sizeof(*h_C_dense));

    // --- Create device arrays and copy host arrays to them
    float *d_A_dense;  gpuErrchk(hipMalloc(&d_A_dense, m * k * sizeof(*d_A_dense)));
    float *d_B_dense;  gpuErrchk(hipMalloc(&d_B_dense, k * n * sizeof(*d_B_dense)));
    float *d_C_dense;  gpuErrchk(hipMalloc(&d_C_dense, m * n * sizeof(*d_C_dense)));
    gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, m * k * sizeof(*d_A_dense), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B_dense, h_B_dense, k * n * sizeof(*d_B_dense), hipMemcpyHostToDevice));

    // --- Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType     (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));

    // --- Descriptor for sparse matrix B
    hipsparseMatDescr_t descrB;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
    cusparseSafeCall(hipsparseSetMatType     (descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));

    // --- Descriptor for sparse matrix C
    hipsparseMatDescr_t descrC;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
    cusparseSafeCall(hipsparseSetMatType     (descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

    int nnzA = 0;                           // --- Number of nonzero elements in dense matrix A
    int nnzB = 0;                           // --- Number of nonzero elements in dense matrix B

    const int lda = m;                      // --- Leading dimension of dense matrix

    // --- Device side number of nonzero elements per row of matrix A
    // TODO: is the size of d_nnzPerVectorA correct?
    int *d_nnzPerVectorA;   gpuErrchk(hipMalloc(&d_nnzPerVectorA, k * sizeof(*d_nnzPerVectorA)));
    cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, m, k, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));

    // --- Device side number of nonzero elements per row of matrix B
    int *d_nnzPerVectorB;   gpuErrchk(hipMalloc(&d_nnzPerVectorB, n * sizeof(*d_nnzPerVectorB)));
    cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, k, n, descrB, d_B_dense, lda, d_nnzPerVectorB, &nnzB));

    // --- Host side number of nonzero elements per row of matrix A
    int *h_nnzPerVectorA = (int *)malloc(k * sizeof(*h_nnzPerVectorA));
    gpuErrchk(hipMemcpy(h_nnzPerVectorA, d_nnzPerVectorA, k * sizeof(*h_nnzPerVectorA), hipMemcpyDeviceToHost));

    // --- Host side number of nonzero elements per row of matrix B
    int *h_nnzPerVectorB = (int *)malloc(n * sizeof(*h_nnzPerVectorB));
    gpuErrchk(hipMemcpy(h_nnzPerVectorB, d_nnzPerVectorB, n * sizeof(*h_nnzPerVectorB), hipMemcpyDeviceToHost));

    // --- Device side sparse matrix
    float *d_A;            gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
    float *d_B;            gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));

    int *d_A_RowIndices;    gpuErrchk(hipMalloc(&d_A_RowIndices, (m + 1) * sizeof(*d_A_RowIndices)));
    int *d_B_RowIndices;    gpuErrchk(hipMalloc(&d_B_RowIndices, (k + 1) * sizeof(*d_B_RowIndices)));
    int *d_C_RowIndices;    gpuErrchk(hipMalloc(&d_C_RowIndices, (m + 1) * sizeof(*d_C_RowIndices)));
    int *d_A_ColIndices;    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
    int *d_B_ColIndices;    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));

    cusparseSafeCall(hipsparseSdense2csr(handle, m, k, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));
    cusparseSafeCall(hipsparseSdense2csr(handle, k, n, descrB, d_B_dense, lda, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

    // --- Host side sparse matrices
    float *h_A = (float *)malloc(nnzA * sizeof(*h_A));
    float *h_B = (float *)malloc(nnzB * sizeof(*h_B));
    int *h_A_RowIndices = (int *)malloc((m + 1) * sizeof(*h_A_RowIndices));
    int *h_A_ColIndices = (int *)malloc(nnzA * sizeof(*h_A_ColIndices));
    int *h_B_RowIndices = (int *)malloc((k + 1) * sizeof(*h_B_RowIndices));
    int *h_B_ColIndices = (int *)malloc(nnzB * sizeof(*h_B_ColIndices));
    int *h_C_RowIndices = (int *)malloc((m + 1) * sizeof(*h_C_RowIndices));
    gpuErrchk(hipMemcpy(h_A, d_A, nnzA * sizeof(*h_A), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (m + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnzA * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_B, d_B, nnzB * sizeof(*h_B), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_B_RowIndices, d_B_RowIndices, (k + 1) * sizeof(*h_B_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_B_ColIndices, d_B_ColIndices, nnzB * sizeof(*h_B_ColIndices), hipMemcpyDeviceToHost));

    // --- Performing the matrix - matrix multiplication
    int baseC, nnzC = 0;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;

    cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrB, nnzB,
                                         d_B_RowIndices, d_B_ColIndices, descrA, nnzA, d_A_RowIndices, d_A_ColIndices, descrC, d_C_RowIndices,
                                         nnzTotalDevHostPtr));
    if (NULL != nnzTotalDevHostPtr) nnzC = *nnzTotalDevHostPtr;
    else {
        gpuErrchk(hipMemcpy(&nnzC,  d_C_RowIndices + m, sizeof(int), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(&baseC, d_C_RowIndices,     sizeof(int), hipMemcpyDeviceToHost));
        nnzC -= baseC;
    }
    int *d_C_ColIndices;    gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));
    float *d_C;            gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(float)));
    float *h_C = (float *)malloc(nnzC * sizeof(*h_C));
    int *h_C_ColIndices = (int *)malloc(nnzC * sizeof(*h_C_ColIndices));
    cusparseSafeCall(hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrB, nnzB,
                                      d_B, d_B_RowIndices, d_B_ColIndices, descrA, nnzA, d_A, d_A_RowIndices, d_A_ColIndices, descrC,
                                      d_C, d_C_RowIndices, d_C_ColIndices));

    cusparseSafeCall(hipsparseScsr2dense(handle, m, n, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, lda));

    gpuErrchk(hipMemcpy(h_C ,           d_C,            nnzC * sizeof(*h_C), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_C_RowIndices, d_C_RowIndices, (m + 1) * sizeof(*h_C_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_C_ColIndices, d_C_ColIndices, nnzC * sizeof(*h_C_ColIndices), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(h_C_dense, d_C_dense, m * n * sizeof(float), hipMemcpyDeviceToHost));

}