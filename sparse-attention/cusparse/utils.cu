#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "utils.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

        case HIPSPARSE_STATUS_SUCCESS:
            return"HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return"HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return"HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return"HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return"HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return"HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return"HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return"HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return"HIPSPARSE_STATUS_ZERO_PIVOT";

        case HIPSPARSE_STATUS_NOT_SUPPORTED:
            return"HIPSPARSE_STATUS_NOT_SUPPORTED";
    }

    return"<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if (HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr,"CUSPARSE error in file '%s', line %d, error %s terminating!", __FILE__, __LINE__, _cusparseGetErrorEnum(err));
        assert(0);
    }
}

