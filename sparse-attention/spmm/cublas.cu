
#include <hip/hip_runtime.h>
#include <hipblas.h>

// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *h_sparse, const float *h_dense, const int m, const int k, const int n) {
    float *d_sparse, *d_dense, *d_res;
    hipMalloc(&d_sparse, m * k * sizeof(float));
    hipMalloc(&d_dense, k * n * sizeof(float));
    hipMalloc(&d_res, m * n * sizeof(float));
    hipMemcpy(d_sparse, h_sparse, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dense, h_dense, k * n * sizeof(float), hipMemcpyHostToDevice);

    // invoke cublas function
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_sparse, lda, d_dense, ldb, beta, d_res, ldc);
    // Destroy the handle
    hipblasDestroy(handle);

    // copy results back to host
}